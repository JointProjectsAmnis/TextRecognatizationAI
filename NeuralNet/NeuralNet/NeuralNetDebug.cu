#include "hip/hip_runtime.h"

#pragma once

#include "NeuralNetDebug.cuh"
#define COUNT_THREADS 16
#define COUNT_BLOCKS(N, TH) (unsigned int)((N + TH - 1) / TH)

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thread>

__global__ void lerpImage(float* image1, float* image2, int sizeX, int sizeY, int channel)
{
    int idX = (threadIdx.x + blockIdx.x * blockDim.x) * channel;
    int idY = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = idX + idY * gridDim.x * blockDim.x * channel;

    if (offset < sizeX * sizeY * channel)
        for (int x = 0; x < channel; x++)
            image1[offset + x] = image1[offset + x] + (image2[offset + x] - image1[offset + x]) * 0.5f;
}

NeuralNetDebug::NeuralNetDebug(int sizeX, int sizeY, int channel)
{
    image = nullptr;
    this->sizeX = sizeX;
    this->sizeY = sizeY;
    this->channel = channel;

    dimX = { COUNT_BLOCKS(sizeX, COUNT_THREADS), COUNT_BLOCKS(sizeY, COUNT_THREADS) };
    dimY = { COUNT_THREADS, COUNT_THREADS };
}

void NeuralNetDebug::forwardPropogation(float* input)
{
    this->image = input;
}

void NeuralNetDebug::backPropogation(float* teather)
{
    lerpImage << < dimX, dimY >> > (image, teather, sizeX, sizeY, channel);
}

void NeuralNetDebug::learnOne(ISample* sample)
{
}

void NeuralNetDebug::learn(SampleData* sampleData, int countGenerations)
{
}

float* NeuralNetDebug::getOutput()
{
    return image;
}

Image* NeuralNetDebug::getImage(Image*& outImage)
{
    if (outImage == nullptr)
        outImage = new Image(image, sizeX, sizeY, channel, false);
    return outImage;
}

int NeuralNetDebug::getCountLayers()
{
    return 1;
}

int NeuralNetDebug::getCountNeurons(int id)
{
    return sizeX * sizeY * channel;
}